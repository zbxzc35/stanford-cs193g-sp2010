
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

int main(void) {

    int num_bits = 16;
    int num_bytes = num_bits * sizeof(int);

    int* device_array = 0;
    int* host_array = 0;
    
    host_array = (int*) malloc(num_bytes);

    hipMalloc((void**)&device_array, num_bytes);
    hipMemset(device_array, 0, num_bytes);
    hipMemcpy(host_array, device_array, num_bytes, 
            hipMemcpyDeviceToHost);
    for (int i = 0; i < 16; i++) {
        std::cout<<host_array[i]<<std::endl;
    }

    free(host_array);
    hipFree(device_array);
    return 0;
}

